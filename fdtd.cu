#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "book.h"
#include "cpu_anim.h"


#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_VOL 1.0f
#define MIN_VOL 0.00001f
#define SPEED 0.25f

struct DataBlock{
    unsigned char *output_bitmap;
    float *dev_EzSrc;
    float *dev_HxSrc;
    float *dev_HySrc;
    float *dev_epsSrc;
    float *dev_muSrc;
    float *dev_sigmaSrc;
    float *dev_sigmastarSrc;
    float *dev_constSrc;
    CPUAnimBitmap *bitmap;
    hipEvent_t start, stop;
    float totalTime;
    float frames;
};

__global__ void copy_const_kernel(float *iptr, const float *cptr){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    if(cptr[offset] != 0){
        iptr[offset] = cptr[offset];
    }
}

__global__ void update_Hx(float *Hx, float *Ez, float *mu, float *epsilon,
                            float *sigma, float *sigma_star, float delta,
                            float deltat, int xdim, int ydim){

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    float mus = mu[offset];
    float sigmamstar = sigma_star[offset];

    float coef1 = (2.0 * mus - sigmamstar * deltat) / (2.0 * mus + sigmamstar * deltat);
    float coef2 = (2 * deltat) / ((2 * mus + sigmamstar * deltat) * delta);

    int top = offset + xdim;
    if(y < ydim -1)
        Hx[offset] = coef1 * Hx[offset] - coef2 * (Ez[top] - Ez[offset]);
    __syncthreads();
}

__global__ void update_Hy(float *Hy, float *Ez, float *mu, float *epsilon,
                            float *sigma, float *sigma_star, float delta,
                            float deltat, int xdim, int ydim){

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    float mus = mu[offset];
    float sigmamstar = sigma_star[offset];

    float coef1 = (2.0 * mus - sigmamstar * deltat) / (2.0 * mus + sigmamstar * deltat);
    float coef2 = (2 * deltat) / ((2 * mus + sigmamstar * deltat) * delta);

    int right = offset + 1;
    if(x < xdim -1)
        Hy[offset] = coef1 * Hy[offset] + coef2 * (Ez[right] - Ez[offset]);
    __syncthreads();
}

__global__ void update_Ez(float *Hy, float *Ez, float *Hx, float *mu, float *epsilon,
                            float *sigma, float *sigma_star, float delta,
                            float deltat, int xdim, int ydim){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    int left=offset - 1;
    int bottom = offset - xdim;

    float sigmam = sigma[offset];
    float eps = epsilon[offset];
    float mus = mu[offset];
    float coef1 = (2.0 * epsi - sigmam * deltat) / (2.0 * eps + sigmam * deltat);
    float coef2 = (2.0 * eps * deltat) / ((2 * eps + sigmam * deltat) * delta);

    if (x > 0 && y > 0 && x<xdim -1 && y < xdim - 1)
        Ez[offset] = coef1 * Ez[offset] + coef2 * ((Hy[offset] - Hy[left]) -
                                        (Hx[offset] - Hx[bottom]));

    __syncthreads();
}

void anim_gpu(DataBlock *d, int ticks){
    HANDLE_ERROR(hipEventRecord(d->start, 0) );
    dim3 blocks(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    CPUAnimBitmap *bitmap = d->bitmap;
    for(int i=0;i<90;i++){
        copy_const_kernel<<<blocks, threads>>>(d->dev_EzSrc, d->dev_constSrc);
        update Hx<<<blocks, threads>>>(d->dev_outSrc, d->dev_inSrc);
        update Hy<<<blocks, threads>>>(d->dev_outSrc, d->dev_inSrc);
        update Ez<<<blocks, threads>>>(d->dev_outSrc, d->dev_inSrc);
        swap(d->dev_inSrc, d->dev_outSrc);
    }
    float_to_color<<<blocks, threads>>> (d->output_bitmap, d->dev_inSrc);
    HANDLE_ERROR(hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipEventRecord(d->stop, 0) );
    HANDLE_ERROR(hipEventSynchronize(d->stop));
    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, d->start, d->stop));
    d->totalTime +=elapsedTime;
    d->frames +=1;
    printf("Average time per frame: %3.1f ms\n", d->totalTime / d->frames);
}

void anim_exit(DataBlock *d){
    hipFree(d->dev_inSrc);
    hipFree(d->dev_outSrc);
    hipFree(d->dev_constSrc);
    HANDLE_ERROR(hipEventDestroy(d->start) );
    HANDLE_ERROR(hipEventDestroy(d->stop) );
}

int main(){
    DataBlock data;
    CPUAnimBitmap bitmap(DIM, DIM, &data);
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    HANDLE_ERROR(hipEventCreate(&data.start, 1) );
    HANDLE_ERROR(hipEventCreate(&data.stop, 1) );

    HANDLE_ERROR(hipMalloc( (void **) &data.output_bitmap, bitmap.image_size() ));
    HANDLE_ERROR(hipMalloc( (void **) &data.dev_inSrc, bitmap.image_size() ));
    HANDLE_ERROR(hipMalloc( (void **) &data.dev_outSrc, bitmap.image_size() ));
    HANDLE_ERROR(hipMalloc( (void **) &data.dev_constSrc, bitmap.image_size() ));

    float *temp = (float *) malloc(bitmap.image_size() );
    int i = 0;

    for (i= 0; i<DIM * DIM; i++){
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if( (x > 300) && (x < 600) && (y> 310) && (y<610) ){
            temp[i] = MAX_VOL;
        }
    }
    HANDLE_ERROR(hipMemcpy(data.dev_constSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(data.dev_inSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice));
    free(temp);
    bitmap.anim_and_exit( (void (*)(void *, int)) anim_gpu, (void (*)(void *)) anim_exit);
}
