
#include <hip/hip_runtime.h>
__device__ __constant__ int x_index_dim;
__device__ __constant__ int y_index_dim;
__device__ __constant__ float delta;
__device__ __constant__ float deltat;

__global__ void copy_const_kernel(float *iptr, const float *cptr){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * x_index_dim;
    if(cptr[offset] != 0){
        iptr[offset] = cptr[offset];
    }
    __syncthreads();
}

__global__ void update_Hx(float *Hx, float *Ez, float *coef1, float* coef2){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * x_index_dim;
    int top = offset + x_index_dim;
    if(y < y_index_dim -1)
        Hx[offset] = coef1[offset] * Hx[offset]
                        - coef2[offset] * (Ez[top] - Ez[offset]);
    __syncthreads();
}

__global__ void update_Hy(float *Hy, float *Ez, float * coef1, float * coef2){

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * x_index_dim;
    int right = offset + 1;
    if(x < x_index_dim -1)
        Hy[offset] = coef1[offset] * Hy[offset] + 
                        coef2[offset] * (Ez[right] - Ez[offset]);
    __syncthreads();
}

__global__ void update_Ez(float *Hx, float *Hy, float *Ez, float * coef1,
                            float *coef2){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    int left = offset - 1;
    int bottom = offset - x_index_dim;

    if (x > 0 && y > 0 && x<x_index_dim - 1 && y < y_index_dim - 1){
        Ez[offset] = coef1[offset] * Ez[offset] +
                    coef2[offset] * ((Hy[offset] - Hy[left]) -
                                    (Hx[offset] - Hx[bottom]));
    }

    __syncthreads();
}



__global__ void te_getcoeff(float *mu,
                                float * epsilon,
                                float *sigma,
                                float * sigma_star,
                                float * coef1,
                                float * coef2,
                                float * coef3,
                                float * coef4){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    float mus = mu[offset];
    float sigmamstar = sigma_star[offset];
    float sigmam = sigma[offset];
    float eps = epsilon[offset];
    coef1[offset] = (2.0 * mus - sigmamstar * deltat) /
                        (2.0 * mus + sigmamstar * deltat);
    coef2[offset] = (2 * deltat) / ((2 * mus + sigmamstar * deltat) * delta);

    coef3[offset] = (2.0 * eps - sigmam * deltat) /
                        (2.0 * eps + sigmam * deltat);
    coef4[offset] = (2.0 * deltat) /
                    ((2 * eps + sigmam * deltat) * delta);
    __syncthreads();
}
