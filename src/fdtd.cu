#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "book.h"
#include "cpu_anim.h"
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
#include "h5save.h"
#include<stdio.h>
#include<pthread.h>
#include "datablock.h"
#include "kernels.cu"
#include "constants.h"


void anim_gpu(Datablock *d, int ticks){
    checkCudaErrors(hipEventRecord(d->start, 0) );
    dim3 blocks((d->structure->x_index_dim + BLOCKSIZE_X - 1) / BLOCKSIZE_X,
                (d->structure->y_index_dim + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y);

    dim3 threads(BLOCKSIZE_X, BLOCKSIZE_Y);
    CPUAnimBitmap *bitmap = d->bitmap;
    for(int i=0;i<100;i++){
        copy_const_kernel<<<blocks, threads>>>(d->fields[TE_EZFIELD],
                                                d->dev_const);

        update_Hx<<<blocks, threads>>>(d->fields[TE_HXFIELD],
                                        d->fields[TE_EZFIELD],
                                        d->coefs[0],
                                        d->coefs[1]);

        update_Hy<<<blocks, threads>>>(d->fields[TE_HYFIELD],
                                        d->fields[TE_EZFIELD],
                                        d->coefs[0],
                                        d->coefs[1]);

        update_Ez<<<blocks, threads>>>(d->fields[TE_HXFIELD],
                                        d->fields[TE_HYFIELD],
                                        d->fields[TE_EZFIELD],
                                        d->coefs[2],
                                        d->coefs[3]);
    }
    float_to_color<<<blocks, threads>>> (d->output_bitmap,
                                        d->fields[TE_EZFIELD]);

    checkCudaErrors(hipMemcpy(bitmap->get_ptr(), d->output_bitmap,
                        bitmap->image_size(), hipMemcpyDeviceToHost));

    checkCudaErrors(hipEventRecord(d->stop, 0) );
    checkCudaErrors(hipEventSynchronize(d->stop));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, d->start, d->stop));
    d->totalTime +=elapsedTime;
    d->frames +=1;
    printf("Average time per frame: %3.1f ms\n", elapsedTime);
}

void anim_exit(Datablock *d){
    hipFree(d->fields[TE_EZFIELD]);
    hipFree(d->fields[TE_HYFIELD]);
    hipFree(d->fields[TE_HXFIELD]);
    hipFree(d->constants[SIGMAINDEX]);
    hipFree(d->constants[SIGMA_STAR_INDEX]);
    hipFree(d->constants[EPSINDEX]);
    hipFree(d->constants[MUINDEX]);
    hipFree(d->coefs[0]);
    hipFree(d->coefs[1]);
    hipFree(d->coefs[2]);
    hipFree(d->coefs[3]);
    hipFree(d->dev_const);
    checkCudaErrors(hipEventDestroy(d->start) );
    checkCudaErrors(hipEventDestroy(d->stop) );
}

void allocateTEMemory(Datablock *data, Structure *structure){
    printf("The size of the structure is %d", structure->size());

    checkCudaErrors(hipMalloc( (void **) &data->output_bitmap,
                    structure->size()));
    checkCudaErrors(hipMalloc( (void **) &data->fields[TE_EZFIELD],
                    structure->size() ));
    checkCudaErrors(hipMalloc( (void **) &data->fields[TE_HYFIELD],
                    structure->size() ));
    checkCudaErrors(hipMalloc( (void **) &data->fields[TE_HXFIELD],
                    structure->size() ));
    checkCudaErrors(hipMalloc( (void **) &data->constants[MUINDEX],
                    structure->size() ));
    checkCudaErrors(hipMalloc( (void **) &data->constants[EPSINDEX],
                    structure->size() ));
    checkCudaErrors(hipMalloc( (void **) &data->constants[SIGMAINDEX],
                    structure->size() ));
    checkCudaErrors(hipMalloc( (void **) &data->constants[SIGMA_STAR_INDEX],
                    structure->size() ));
    checkCudaErrors(hipMalloc( (void **) &data->dev_const,
                    structure->size() ));
    checkCudaErrors(hipMalloc( (void **) &data->coefs[0],
                    structure->size() ));
    checkCudaErrors(hipMalloc( (void **) &data->coefs[1],
                    structure->size() ));
    checkCudaErrors(hipMalloc( (void **) &data->coefs[2],
                    structure->size() ));
    checkCudaErrors(hipMalloc( (void **) &data->coefs[3],
                    structure->size() ));

}

int main(){
    Datablock data(TE_SIMULATION);
    float dt= 0.5;
// FIXME: check the courant factor for the max epsilon.
    float courant = 0.5;
    float dx =  (dt * LIGHTSPEED) / courant;
    Structure structure(1024, 1024, dx, dt);


    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(x_index_dim), &structure.x_index_dim,
                    sizeof(structure.x_index_dim)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(y_index_dim), &structure.y_index_dim,
                    sizeof(structure.y_index_dim)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(delta), &structure.dx,
                    sizeof(structure.dx)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(deltat), &structure.dt,
                    sizeof(structure.dt)));
    CPUAnimBitmap bitmap(structure.x_index_dim, structure.x_index_dim,
                            &data);

    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    data.structure = &structure;
    checkCudaErrors(hipEventCreate(&data.start, 1) );
    checkCudaErrors(hipEventCreate(&data.stop, 1) );
    allocateTEMemory(&data, &structure);


    float *temp = (float *) malloc(bitmap.image_size() );
    float *temp_mu = (float *) malloc(bitmap.image_size() );
    for(int i=0;i<structure.x_index_dim;i++)
        for(int j=0;j<structure.y_index_dim;j++)
            temp_mu[i + j * structure.x_index_dim] = MU;

    checkCudaErrors(hipMemcpy(data.constants[MUINDEX], temp_mu, bitmap.image_size(),
                    hipMemcpyHostToDevice));

    for(int i=0;i<structure.x_index_dim;i++)
        for(int j=0;j<structure.y_index_dim;j++)
            temp_mu[i + j * structure.x_index_dim] = EPSILON * 20;

    checkCudaErrors(hipMemcpy(data.constants[EPSINDEX], temp_mu, bitmap.image_size(),
                    hipMemcpyHostToDevice));

    for(int i=0;i<structure.x_index_dim;i++)
        for(int j=0;j<structure.y_index_dim;j++)
            temp_mu[i + j * structure.x_index_dim] = 0;

    for(int i=0;i<structure.x_index_dim;i++)
        for(int j=0;j<structure.y_index_dim;j++)
            temp[i + j * structure.x_index_dim] = 0;

    checkCudaErrors(hipMemcpy(data.constants[SIGMAINDEX], temp_mu, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(data.constants[SIGMA_STAR_INDEX], temp_mu, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(data.fields[TE_EZFIELD], temp, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(data.fields[TE_HXFIELD], temp, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(data.fields[TE_HYFIELD], temp, bitmap.image_size(),
                    hipMemcpyHostToDevice));

//  get the coefficients

    dim3 blocks((structure.x_index_dim + BLOCKSIZE_X - 1) / BLOCKSIZE_X,
                (structure.y_index_dim + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y);
    dim3 threads(BLOCKSIZE_X, BLOCKSIZE_Y);

    te_getcoeff<<<blocks, threads>>>(data.constants[0],
                                     data.constants[1],
                                     data.constants[2],
                                     data.constants[3],
                                     data.coefs[0],
                                     data.coefs[1],
                                     data.coefs[2],
                                     data.coefs[3]);
    hipFree(data.constants[0]);
    hipFree(data.constants[1]);
    hipFree(data.constants[2]);
    hipFree(data.constants[3]);


    for(int i= 125; i< 129;i++)
        for(int j=125; j<129;j++)
        temp[256 * j + i] = 1;

    checkCudaErrors(hipMemcpy(data.dev_const, temp, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    free(temp);
    free(temp_mu);
    bitmap.anim_and_exit( (void (*)(void *, int)) anim_gpu,
                            (void (*)(void *)) anim_exit);
}
