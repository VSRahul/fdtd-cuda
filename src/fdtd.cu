#include "hip/hip_runtime.h"
#include "book.h"
#include "cpu_anim.h"
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
#include "h5save.h"
#include<stdio.h>
#include<pthread.h>
#include "datablock.h"
#include "kernels.cu"


#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_VOL 1.0f
#define MIN_VOL 0.00001f
#define LIGHTSPEED 299792458
#define EPSILON 8.8541878176e-12f
#define MU 1.2566370614e-6f





void anim_gpu(Datablock *d, int ticks){
    checkCudaErrors(hipEventRecord(d->start, 0) );
    dim3 blocks(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    CPUAnimBitmap *bitmap = d->bitmap;
    for(int i=0;i<100;i++){
        copy_const_kernel<<<blocks, threads>>>(d->fields[TE_EZFIELD],
                                                d->dev_const);

        update_Hx<<<blocks, threads>>>(d->fields[TE_HXFIELD],
                                        d->fields[TE_EZFIELD],
                                        d->constants[SIGMA_STAR_INDEX],
                                        d->constants[MUINDEX]);

        update_Hy<<<blocks, threads>>>(d->fields[TE_HYFIELD],
                                        d->fields[TE_EZFIELD],
                                        d->constants[SIGMA_STAR_INDEX],
                                        d->constants[MUINDEX]);

        update_Ez<<<blocks, threads>>>(d->fields[TE_HXFIELD],
                                        d->fields[TE_HYFIELD],
                                        d->fields[TE_EZFIELD],
                                        d->constants[SIGMAINDEX],
                                        d->constants[EPSINDEX]);
    }
    float_to_color<<<blocks, threads>>> (d->output_bitmap,
                                        d->fields[TE_EZFIELD]);

    checkCudaErrors(hipMemcpy(bitmap->get_ptr(), d->output_bitmap,
                        bitmap->image_size(), hipMemcpyDeviceToHost));

    checkCudaErrors(hipEventRecord(d->stop, 0) );
    checkCudaErrors(hipEventSynchronize(d->stop));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, d->start, d->stop));
    d->totalTime +=elapsedTime;
    d->frames +=1;
    printf("Average time per frame: %3.1f ms\n", d->totalTime / d->frames);
}

void anim_exit(Datablock *d){
    hipFree(d->fields[TE_EZFIELD]);
    hipFree(d->fields[TE_HYFIELD]);
    hipFree(d->fields[TE_HXFIELD]);
    hipFree(d->constants[SIGMAINDEX]);
    hipFree(d->constants[SIGMA_STAR_INDEX]);
    hipFree(d->constants[EPSINDEX]);
    hipFree(d->constants[MUINDEX]);
    checkCudaErrors(hipEventDestroy(d->start) );
    checkCudaErrors(hipEventDestroy(d->stop) );
}

int main(){
    Datablock data(0);
    Structure structure;
    structure.x_index_dim = 1024;
    structure.y_index_dim = 1024;
    structure.dt= 0.5;
    structure.courant = 0.5;
    structure.dx =  (structure.dt * LIGHTSPEED) / structure.courant; 
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(x_index_dim), &structure.x_index_dim,
                    sizeof(structure.x_index_dim)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(y_index_dim), &structure.y_index_dim,
                    sizeof(structure.y_index_dim)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(delta), &structure.dx,
                    sizeof(structure.dx)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(deltat), &structure.dt,
                    sizeof(structure.dt)));
    CPUAnimBitmap bitmap(structure.x_index_dim, structure.x_index_dim,
                            &data);
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    checkCudaErrors(hipEventCreate(&data.start, 1) );
    checkCudaErrors(hipEventCreate(&data.stop, 1) );

    checkCudaErrors(hipMalloc( (void **) &data.output_bitmap,
                    bitmap.image_size()));
    checkCudaErrors(hipMalloc( (void **) &data.fields[TE_EZFIELD], bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.fields[TE_HYFIELD], bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.fields[TE_HXFIELD], bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.constants[MUINDEX], bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.constants[EPSINDEX], bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.constants[SIGMAINDEX], bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.constants[SIGMA_STAR_INDEX], bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.dev_const, bitmap.image_size() ));

    float *temp = (float *) malloc(bitmap.image_size() );
    float *temp_mu = (float *) malloc(bitmap.image_size() );
    for(int i=0;i<structure.x_index_dim;i++)
        for(int j=0;j<structure.y_index_dim;j++)
            temp_mu[i + j * structure.x_index_dim] = MU;

    checkCudaErrors(hipMemcpy(data.constants[MUINDEX], temp_mu, bitmap.image_size(),
                    hipMemcpyHostToDevice));

    for(int i=0;i<structure.x_index_dim;i++)
        for(int j=0;j<structure.y_index_dim;j++)
            temp_mu[i + j * structure.x_index_dim] = EPSILON;

    checkCudaErrors(hipMemcpy(data.constants[EPSINDEX], temp_mu, bitmap.image_size(),
                    hipMemcpyHostToDevice));

    for(int i=0;i<structure.x_index_dim;i++)
        for(int j=0;j<structure.y_index_dim;j++)
            temp_mu[i + j * structure.x_index_dim] = 0;

    for(int i=0;i<structure.x_index_dim;i++)
        for(int j=0;j<structure.y_index_dim;j++)
            temp[i + j * structure.x_index_dim] = 0;
    checkCudaErrors(hipMemcpy(data.constants[SIGMAINDEX], temp_mu, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(data.constants[SIGMA_STAR_INDEX], temp_mu, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(data.fields[TE_EZFIELD], temp, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(data.fields[TE_HXFIELD], temp, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(data.fields[TE_HYFIELD], temp, bitmap.image_size(),
                    hipMemcpyHostToDevice));

    for(int i= 125; i< 129;i++)
        for(int j=125; j<129;j++)
        temp[256 * j + i] = 1;

    checkCudaErrors(hipMemcpy(data.dev_const, temp, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    free(temp);
    free(temp_mu);
    bitmap.anim_and_exit( (void (*)(void *, int)) anim_gpu,
                            (void (*)(void *)) anim_exit);
}
