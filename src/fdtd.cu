#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "book.h"
#include "cpu_anim.h"
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
#include "h5save.h"
#include<stdio.h>
#include<pthread.h>
#include "datablock.h"
#include "kernels.cu"
#include "constants.h"

#define DIM 1024


void anim_gpu(Datablock *d, int ticks){
    checkCudaErrors(hipEventRecord(d->start, 0) );
    dim3 blocks((d->structure->x_index_dim + BLOCKSIZE_X - 1) / BLOCKSIZE_X,
                (d->structure->y_index_dim + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y);

    dim3 threads(BLOCKSIZE_X, BLOCKSIZE_Y);

    dim3 blocksnew((d->structure->x_index_dim + BLOCKSIZE_HX - 1) / BLOCKSIZE_HX,
                (d->structure->y_index_dim + BLOCKSIZE_HY - 1) / BLOCKSIZE_HY);

    dim3 threadsnew(BLOCKSIZE_HX, BLOCKSIZE_HY);
    CPUAnimBitmap *bitmap = d->bitmap;
    for(int i=0;i<100;i++){
        copy_const_kernel<<<blocks, threads>>>(d->fields[TE_EZFIELD],
                                                d->dev_const);

        update_Hx<<<blocksnew, threadsnew>>>(d->fields[TE_HXFIELD],
                                        d->fields[TE_EZFIELD],
                                        d->coefs[0],
                                        d->coefs[1]);

        update_Hy<<<blocks, threads>>>(d->fields[TE_HYFIELD],
                                        d->fields[TE_EZFIELD],
                                        d->coefs[0],
                                        d->coefs[1]);

        update_Ez<<<blocks, threads>>>(d->fields[TE_HXFIELD],
                                        d->fields[TE_HYFIELD],
                                        d->fields[TE_EZFIELD],
                                        d->coefs[2],
                                        d->coefs[3]);
    }
    float_to_color<<<blocks, threads>>> (d->output_bitmap,
                                        d->fields[TE_EZFIELD]);

    checkCudaErrors(hipMemcpy(bitmap->get_ptr(), d->output_bitmap,
                        bitmap->image_size(), hipMemcpyDeviceToHost));

    checkCudaErrors(hipEventRecord(d->stop, 0) );
    checkCudaErrors(hipEventSynchronize(d->stop));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, d->start, d->stop));
    d->totalTime +=elapsedTime;
    d->frames +=1;
    printf("Average time per frame: %3.1f ms\n", elapsedTime);
}

void anim_exit(Datablock *d){
    hipFree(d->fields[TE_EZFIELD]);
    hipFree(d->fields[TE_HYFIELD]);
    hipFree(d->fields[TE_HXFIELD]);
    hipFree(d->constants[SIGMAINDEX]);
    hipFree(d->constants[SIGMA_STAR_INDEX]);
    hipFree(d->constants[EPSINDEX]);
    hipFree(d->constants[MUINDEX]);
    hipFree(d->coefs[0]);
    hipFree(d->coefs[1]);
    hipFree(d->coefs[2]);
    hipFree(d->coefs[3]);
    hipFree(d->dev_const);
    checkCudaErrors(hipEventDestroy(d->start) );
    checkCudaErrors(hipEventDestroy(d->stop) );
}

int main(){
    Datablock data(TE_SIMULATION);
    Structure structure;
    structure.x_index_dim = 1024;
    structure.y_index_dim = 1024;
    structure.dt= 0.5;
    structure.courant = 0.5;

// FIXME: check the courant factor for the max epsilon.

    structure.dx =  (structure.dt * LIGHTSPEED) / structure.courant;
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(x_index_dim), &structure.x_index_dim,
                    sizeof(structure.x_index_dim)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(y_index_dim), &structure.y_index_dim,
                    sizeof(structure.y_index_dim)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(delta), &structure.dx,
                    sizeof(structure.dx)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(deltat), &structure.dt,
                    sizeof(structure.dt)));
    CPUAnimBitmap bitmap(structure.x_index_dim, structure.x_index_dim,
                            &data);

    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    data.structure = &structure;
    checkCudaErrors(hipEventCreate(&data.start, 1) );
    checkCudaErrors(hipEventCreate(&data.stop, 1) );

    checkCudaErrors(hipMalloc( (void **) &data.output_bitmap,
                    bitmap.image_size()));
    checkCudaErrors(hipMalloc( (void **) &data.fields[TE_EZFIELD],
                    bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.fields[TE_HYFIELD],
                    bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.fields[TE_HXFIELD],
                    bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.constants[MUINDEX],
                    bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.constants[EPSINDEX],
                    bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.constants[SIGMAINDEX],
                    bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.constants[SIGMA_STAR_INDEX],
                    bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.dev_const,
                    bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.coefs[0],
                    bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.coefs[1],
                    bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.coefs[2],
                    bitmap.image_size() ));
    checkCudaErrors(hipMalloc( (void **) &data.coefs[3],
                    bitmap.image_size() ));


    float *temp = (float *) malloc(bitmap.image_size() );
    float *temp_mu = (float *) malloc(bitmap.image_size() );
    for(int i=0;i<structure.x_index_dim;i++)
        for(int j=0;j<structure.y_index_dim;j++)
            temp_mu[i + j * structure.x_index_dim] = MU;

    checkCudaErrors(hipMemcpy(data.constants[MUINDEX], temp_mu, bitmap.image_size(),
                    hipMemcpyHostToDevice));

    for(int i=0;i<structure.x_index_dim;i++)
        for(int j=0;j<structure.y_index_dim;j++)
            temp_mu[i + j * structure.x_index_dim] = EPSILON * 20;

    checkCudaErrors(hipMemcpy(data.constants[EPSINDEX], temp_mu, bitmap.image_size(),
                    hipMemcpyHostToDevice));

    for(int i=0;i<structure.x_index_dim;i++)
        for(int j=0;j<structure.y_index_dim;j++)
            temp_mu[i + j * structure.x_index_dim] = 0;

    for(int i=0;i<structure.x_index_dim;i++)
        for(int j=0;j<structure.y_index_dim;j++)
            temp[i + j * structure.x_index_dim] = 0;

    checkCudaErrors(hipMemcpy(data.constants[SIGMAINDEX], temp_mu, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(data.constants[SIGMA_STAR_INDEX], temp_mu, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(data.fields[TE_EZFIELD], temp, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(data.fields[TE_HXFIELD], temp, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(data.fields[TE_HYFIELD], temp, bitmap.image_size(),
                    hipMemcpyHostToDevice));

//  get the coefficients
// FIXME: Use of DIM

    dim3 blocks((structure.x_index_dim + BLOCKSIZE_X - 1) / BLOCKSIZE_X,
                (structure.y_index_dim + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y);
    dim3 threads(BLOCKSIZE_X, BLOCKSIZE_Y);

    te_getcoeff<<<blocks, threads>>>(data.constants[0],
                                     data.constants[1],
                                     data.constants[2],
                                     data.constants[3],
                                     data.coefs[0],
                                     data.coefs[1],
                                     data.coefs[2],
                                     data.coefs[3]);
    hipFree(data.constants[0]);
    hipFree(data.constants[1]);
    hipFree(data.constants[2]);
    hipFree(data.constants[3]);


    for(int i= 125; i< 129;i++)
        for(int j=125; j<129;j++)
        temp[256 * j + i] = 1;

    checkCudaErrors(hipMemcpy(data.dev_const, temp, bitmap.image_size(),
                    hipMemcpyHostToDevice));
    free(temp);
    free(temp_mu);
    bitmap.anim_and_exit( (void (*)(void *, int)) anim_gpu,
                            (void (*)(void *)) anim_exit);
}
