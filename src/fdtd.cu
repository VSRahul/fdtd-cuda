#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cpu_anim.h"
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
#include "h5save.h"
#include<stdio.h>
#include<pthread.h>
#include "datablock.h"
#include "kernels.cuh"
#include "constants.h"
#include <thrust/fill.h>
#include<algorithm>
#include "tm_mode.h"
#include "pml_mode.h"
#include "drude_mode.h"

void anim_gpu(Datablock *d, int ticks){
    if(d->simulationType == TM_SIMULATION)
        anim_gpu_tm(d, ticks);
    else if(d->simulationType == TM_PML_SIMULATION)
        anim_gpu_pml_tm(d, ticks);
    else if(d->simulationType == DRUDE_SIMULATION)
        anim_gpu_drude(d, ticks);
}

void anim_exit(Datablock *d){
    if(d->simulationType == TM_SIMULATION)
        clear_memory_TM_simulation(d);
    else if(d->simulationType == TM_PML_SIMULATION)
        clear_memory_TM_PML_simulation(d);
    else if(d->simulationType == DRUDE_SIMULATION)
        clear_memory_drude_simulation(d);

}

// FIXME: Fix all uses of pitch here
size_t allocate_memory(Datablock *data, Structure structure){
    if(data->simulationType == TM_SIMULATION)
        return allocateTMMemory(data, structure);
    else if(data->simulationType == TM_PML_SIMULATION)
        tm_pml_allocate_memory(data, structure);
    else if(data->simulationType == DRUDE_SIMULATION)
        allocate_drude_memory(data, structure);
    return 0;
}

void initializeArrays(Datablock *data, Structure structure){
    if(data->simulationType == TM_SIMULATION)
        initialize_TM_arrays(data, structure);
    else if(data->simulationType == TM_PML_SIMULATION)
        tm_pml_initialize_arrays(data, structure);
    else if(data->simulationType == DRUDE_SIMULATION)
        initialize_drude_arrays(data, structure);
}

void clear_memory_constants(Datablock *data){
    if(data->simulationType == TM_SIMULATION)
        tm_clear_memory_constants(data);
    else if(data->simulationType == TM_PML_SIMULATION)
        tm_pml_clear_memory_constants(data);
    else if(data->simulationType == DRUDE_SIMULATION)
        drude_clear_memory_constants(data);

}


void calculate_coefficients(Datablock *data, Structure structure){
    dim3 blocks((structure.x_index_dim + BLOCKSIZE_X - 1) / BLOCKSIZE_X,
                (structure.y_index_dim + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y);
    dim3 threads(BLOCKSIZE_X, BLOCKSIZE_Y);
    if(data->simulationType == TM_SIMULATION)
        tm_getcoeff<<<blocks, threads>>>(data->constants[MUINDEX],
                                         data->constants[EPSINDEX],
                                         data->constants[SIGMAINDEX],
                                         data->constants[SIGMA_STAR_INDEX],
                                         data->coefs[0],
                                         data->coefs[1],
                                         data->coefs[2],
                                         data->coefs[3]
                                         );

    else if(data->simulationType == TM_PML_SIMULATION)
        pml_tm_get_coefs<<<blocks, threads>>>(data->constants[MUINDEX],
                                              data->constants[EPSINDEX],
                                              data->constants[SIGMAINDEX_X],
                                              data->constants[SIGMAINDEX_Y],
                                              data->constants[SIGMA_STAR_INDEX_X],
                                              data->constants[SIGMA_STAR_INDEX_Y],
                                              data->coefs[0],
                                              data->coefs[1],
                                              data->coefs[2],
                                              data->coefs[3],
                                              data->coefs[4],
                                              data->coefs[5],
                                              data->coefs[6],
                                              data->coefs[7]);

    else if(data->simulationType == DRUDE_SIMULATION)
        drude_get_coefs<<<blocks, threads>>>(data->constants[MUINDEX],
                                         data->constants[EPSINDEX],
                                         data->constants[SIGMAINDEX],
                                         data->constants[SIGMA_STAR_INDEX],
                                         data->constants[GAMMA_INDEX],
                                         data->constants[OMEGAP_INDEX],
                                         data->coefs[0],
                                         data->coefs[1],
                                         data->coefs[2],
                                         data->coefs[3],
                                         data->coefs[4],
                                         data->coefs[5],
                                         data->coefs[6]
                                         );

}

int main(){
    Datablock data(TM_SIMULATION);
    float dx= 1e-6 / 300.0;

// FIXME: check the courant factor for the max epsilon.

    float courant = 0.5;
    float dt =  courant * dx / LIGHTSPEED;
    printf("dt = %f", dt);
    Structure structure(1024, 1024, dx, dt);


    CPUAnimBitmap bitmap(structure.x_index_dim, structure.x_index_dim,
                            &data);

    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    data.structure = &structure;
    checkCudaErrors(hipEventCreate(&data.start, 1) );
    checkCudaErrors(hipEventCreate(&data.stop, 1) );

    size_t pitch;
    pitch = allocate_memory(&data, structure);
    structure.pitch = pitch;
    copy_symbols(&structure);
    printf("pitch = %d", pitch);
    initializeArrays(&data, structure);


//  get the coefficients
    calculate_coefficients(&data, structure);


clear_memory_constants(&data);


// set the sources
    HostSources host_sources;
    DeviceSources device_sources;
    host_sources.add_source(16, 16, SINUSOID_SOURCE, 2 * PI * 5e14, 1);

    data.sources = &device_sources;
    copy_sources_device_to_host(&host_sources, &device_sources);

    bitmap.anim_and_exit( (void (*)(void *, int)) anim_gpu,
                            (void (*)(void *)) anim_exit);
}
