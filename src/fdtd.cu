#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cpu_anim.h"
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
#include "h5save.h"
#include<stdio.h>
#include<pthread.h>
#include "datablock.h"
#include "kernels.cuh"
#include "constants.h"
#include <thrust/fill.h>
#include<algorithm>
#include "tm_mode.h"


void anim_gpu(Datablock *d, int ticks){
    if(d->simulationType == TM_SIMULATION)
        anim_gpu_tm(d, ticks);
}


void anim_exit(Datablock *d){
    if(d->simulationType == TM_SIMULATION)
        clear_memory_TM_simulation(d);

}
void allocate_memory(Datablock *data, Structure structure){
    if(data->simulationType == TM_SIMULATION)
        allocateTMMemory(data, structure);
}

void initializeArrays(Datablock *data, Structure structure){
    if(data->simulationType == TM_SIMULATION)
        initialize_TM_arrays(data, structure);
}

void copy_sources(HostSources * host_sources, DeviceSources *device_sources){
    int number_of_sources = host_sources->get_size();
    device_sources->size = number_of_sources;
    checkCudaErrors(hipMalloc((void**)&device_sources->x_source_position,
                number_of_sources * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&device_sources->y_source_position,
                number_of_sources * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&device_sources->source_type,
                number_of_sources * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&device_sources->mean,
                number_of_sources * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&device_sources->variance,
                number_of_sources * sizeof(float)));

    if(number_of_sources != 0){
    int *host_source_ptr = &(host_sources->x_source_position[0]);
    checkCudaErrors(hipMemcpy(device_sources->x_source_position, host_source_ptr,
                sizeof(int) * number_of_sources,hipMemcpyHostToDevice));


    host_source_ptr = &(host_sources->y_source_position[0]);
    checkCudaErrors(hipMemcpy(device_sources->y_source_position, host_source_ptr,
                sizeof(int) * number_of_sources,hipMemcpyHostToDevice));


    host_source_ptr = &(host_sources->source_type[0]);
    checkCudaErrors(hipMemcpy(device_sources->source_type, host_source_ptr,
                sizeof(int) * number_of_sources, hipMemcpyHostToDevice));

    float * mean_ptr = &(host_sources->mean[0]);
    checkCudaErrors(hipMemcpy(device_sources->mean, mean_ptr,
                sizeof(float) * number_of_sources, hipMemcpyHostToDevice));

    float * variance_ptr = &(host_sources->variance[0]);
    checkCudaErrors(hipMemcpy(device_sources->variance, variance_ptr,
                sizeof(float) * number_of_sources, hipMemcpyHostToDevice));
    }
}

int main(){
    Datablock data(TM_SIMULATION);
    float dt= 0.5;
// FIXME: check the courant factor for the max epsilon.
    float courant = 0.5;
    float dx =  (dt * LIGHTSPEED) / courant;
    Structure structure(1024, 1024, dx, dt);
    copy_symbols(&structure);


    CPUAnimBitmap bitmap(structure.x_index_dim, structure.x_index_dim,
                            &data);

    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    data.structure = &structure;
    checkCudaErrors(hipEventCreate(&data.start, 1) );
    checkCudaErrors(hipEventCreate(&data.stop, 1) );

    allocate_memory(&data, structure);
    initializeArrays(&data, structure);


//  get the coefficients

    dim3 blocks((structure.x_index_dim + BLOCKSIZE_X - 1) / BLOCKSIZE_X,
                (structure.y_index_dim + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y);
    dim3 threads(BLOCKSIZE_X, BLOCKSIZE_Y);

    tm_getcoeff<<<blocks, threads>>>(data.constants[0],
                                     data.constants[1],
                                     data.constants[2],
                                     data.constants[3],
                                     data.coefs[0],
                                     data.coefs[1],
                                     data.coefs[2],
                                     data.coefs[3]);

    hipFree(data.constants[0]);
    hipFree(data.constants[1]);
    hipFree(data.constants[2]);
    hipFree(data.constants[3]);

// set the sources
    HostSources host_sources;
    DeviceSources device_sources;
    host_sources.add_source(512, 512, SINUSOID_SOURCE, 0.05, 1);
    host_sources.add_source(256, 512, SINUSOID_SOURCE, 0.1, 1);
    host_sources.add_source(1, 0, SINUSOID_SOURCE, 0.1, 1);

    data.sources = &device_sources;
    copy_sources(&host_sources, &device_sources);

    bitmap.anim_and_exit( (void (*)(void *, int)) anim_gpu,
                            (void (*)(void *)) anim_exit);
}
