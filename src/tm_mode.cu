#include "tm_mode.h"
#include "constants.h"

void anim_gpu_tm(Datablock *d, int ticks){
    checkCudaErrors(hipEventRecord(d->start, 0) );
    dim3 blocks((d->structure->x_index_dim + BLOCKSIZE_X - 1) / BLOCKSIZE_X,
                (d->structure->y_index_dim + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y);
    dim3 threads(BLOCKSIZE_X, BLOCKSIZE_Y);

    dim3 source_threads(64, 1);
    dim3 source_blocks((d->sources->size + 63) / 64, 1);

    CPUAnimBitmap *bitmap = d->bitmap;
    static long time_ticks = 0;
    printf("time ticks = %ld", time_ticks);
    printf("time ticks = %ld", time_ticks);
    for(int i=0;i<100;i++){
        time_ticks += 1;
        copy_sources<<<source_blocks, source_threads>>>(
                d->fields[TM_EZFIELD],
                d->sources->x_source_position,
                d->sources->y_source_position,
                d->sources->source_type,
                d->sources->mean,
                d->sources->variance,
                d->sources->size,
                time_ticks);

        update_Hx<<<blocks, threads>>>(d->fields[TM_HXFIELD],
                                        d->fields[TM_EZFIELD],
                                        d->coefs[0],
                                        d->coefs[1]);

        update_Hy<<<blocks, threads>>>(d->fields[TM_HYFIELD],
                                        d->fields[TM_EZFIELD],
                                        d->coefs[0],
                                        d->coefs[1]);

        update_Ez<<<blocks, threads>>>(d->fields[TM_HXFIELD],
                                        d->fields[TM_HYFIELD],
                                        d->fields[TM_EZFIELD],
                                        d->coefs[2],
                                        d->coefs[3]);
    }
    float_to_color<<<blocks, threads>>> (d->output_bitmap,
                                        d->fields[TM_EZFIELD]);

    checkCudaErrors(hipMemcpy(bitmap->get_ptr(), d->output_bitmap,
                        bitmap->image_size(), hipMemcpyDeviceToHost));

    checkCudaErrors(hipEventRecord(d->stop, 0) );
    checkCudaErrors(hipEventSynchronize(d->stop));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, d->start, d->stop));
    d->totalTime +=elapsedTime;
    d->frames +=1;
    printf("Average time per frame: %3.1f ms\n", elapsedTime);
}


void tm_clear_memory_constants(Datablock *d){
    hipFree(d->constants[SIGMAINDEX]);
    hipFree(d->constants[SIGMA_STAR_INDEX]);
    hipFree(d->constants[EPSINDEX]);
    hipFree(d->constants[MUINDEX]);
}

void clear_memory_TM_simulation(Datablock *d){
    hipFree(d->fields[TM_EZFIELD]);
    hipFree(d->fields[TM_HYFIELD]);
    hipFree(d->fields[TM_HXFIELD]);
    hipFree(d->coefs[0]);
    hipFree(d->coefs[1]);
    hipFree(d->coefs[2]);
    hipFree(d->coefs[3]);
    hipFree(d->sources->x_source_position);
    hipFree(d->sources->y_source_position);
    hipFree(d->sources->source_type);
    hipFree(d->sources->mean);
    hipFree(d->sources->variance);
    checkCudaErrors(hipEventDestroy(d->start) );
    checkCudaErrors(hipEventDestroy(d->stop) );
}

void allocateTMMemory(Datablock *data, Structure structure){
    printf("The size of the structure is %ld", structure.size());

    checkCudaErrors(hipMalloc( (void **) &data->output_bitmap,
                    structure.size()));
    checkCudaErrors(hipMalloc( (void **) &data->fields[TM_EZFIELD],
                    structure.size() ));
    checkCudaErrors(hipMalloc( (void **) &data->fields[TM_HYFIELD],
                    structure.size() ));
    checkCudaErrors(hipMalloc( (void **) &data->fields[TM_HXFIELD],
                    structure.size() ));
    checkCudaErrors(hipMalloc( (void **) &data->constants[MUINDEX],
                    structure.size() ));
    checkCudaErrors(hipMalloc( (void **) &data->constants[EPSINDEX],
                    structure.size() ));
    checkCudaErrors(hipMalloc( (void **) &data->constants[SIGMAINDEX],
                    structure.size() ));
    checkCudaErrors(hipMalloc( (void **) &data->constants[SIGMA_STAR_INDEX],
                    structure.size() ));
    checkCudaErrors(hipMalloc( (void **) &data->coefs[0],
                    structure.size() ));
    checkCudaErrors(hipMalloc( (void **) &data->coefs[1],
                    structure.size() ));
    checkCudaErrors(hipMalloc( (void **) &data->coefs[2],
                    structure.size() ));
    checkCudaErrors(hipMalloc( (void **) &data->coefs[3],
                    structure.size() ));
}

void initialize_TM_arrays(Datablock *data, Structure structure){
    int size = structure.grid_size();
    printf("%ld\n", size);
    printf("%ld\n", structure.x_index_dim);
    printf("%ld\n", structure.y_index_dim);

    // FIXME: Temporary fix for populating values.

    float * temp = (float *) malloc(structure.size());
    std::fill_n(temp, size, MU);
    hipMemcpy(data->constants[MUINDEX],temp,structure.size(),
                hipMemcpyHostToDevice);

    std::fill_n(temp, size, EPSILON * 20);
    hipMemcpy(data->constants[EPSINDEX],temp,structure.size(),
                hipMemcpyHostToDevice);

    std::fill_n(temp, size, 0.0);
    hipMemcpy(data->constants[SIGMAINDEX],temp,structure.size(),
                hipMemcpyHostToDevice);

    std::fill_n(temp, size, 0.0);
    hipMemcpy(data->constants[SIGMA_STAR_INDEX],temp,structure.size(),
                hipMemcpyHostToDevice);

    thrust::device_ptr<float> hx_field_ptr(data->fields[TM_HXFIELD]);
    thrust::fill(hx_field_ptr, hx_field_ptr + size, 0);

    thrust::device_ptr<float> hy_field_ptr(data->fields[TM_HYFIELD]);
    thrust::fill(hy_field_ptr, hy_field_ptr + size, 0);

    thrust::device_ptr<float> ez_field_ptr(data->fields[TM_EZFIELD]);
    thrust::fill(ez_field_ptr, ez_field_ptr + size, 0);

}

void copy_sources_device_to_host(HostSources * host_sources, DeviceSources *device_sources){
    int number_of_sources = host_sources->get_size();
    device_sources->size = number_of_sources;
    checkCudaErrors(hipMalloc((void**)&device_sources->x_source_position,
                number_of_sources * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&device_sources->y_source_position,
                number_of_sources * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&device_sources->source_type,
                number_of_sources * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&device_sources->mean,
                number_of_sources * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&device_sources->variance,
                number_of_sources * sizeof(float)));

    if(number_of_sources != 0){
    int *host_source_ptr = &(host_sources->x_source_position[0]);
    checkCudaErrors(hipMemcpy(device_sources->x_source_position, host_source_ptr,
                sizeof(int) * number_of_sources,hipMemcpyHostToDevice));


    host_source_ptr = &(host_sources->y_source_position[0]);
    checkCudaErrors(hipMemcpy(device_sources->y_source_position, host_source_ptr,
                sizeof(int) * number_of_sources,hipMemcpyHostToDevice));


    host_source_ptr = &(host_sources->source_type[0]);
    checkCudaErrors(hipMemcpy(device_sources->source_type, host_source_ptr,
                sizeof(int) * number_of_sources, hipMemcpyHostToDevice));

    float * mean_ptr = &(host_sources->mean[0]);
    checkCudaErrors(hipMemcpy(device_sources->mean, mean_ptr,
                sizeof(float) * number_of_sources, hipMemcpyHostToDevice));

    float * variance_ptr = &(host_sources->variance[0]);
    checkCudaErrors(hipMemcpy(device_sources->variance, variance_ptr,
                sizeof(float) * number_of_sources, hipMemcpyHostToDevice));
    }
}
