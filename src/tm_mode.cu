#include "tm_mode.h"
#include "constants.h"

void anim_gpu_tm(Datablock *d, int ticks){
    checkCudaErrors(hipEventRecord(d->start, 0) );
    dim3 blocks((d->structure->x_index_dim + BLOCKSIZE_X - 1) / BLOCKSIZE_X,
                (d->structure->y_index_dim + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y);
    dim3 threads(BLOCKSIZE_X, BLOCKSIZE_Y);

    dim3 source_threads(64, 1);
    dim3 source_blocks((d->sources->size + 63) / 64, 1);

    CPUAnimBitmap *bitmap = d->bitmap;
    static long time_ticks = 0;
    printf("time ticks = %ld", time_ticks);
    printf("time ticks = %ld", time_ticks);
    for(int i=0;i<100;i++){
        time_ticks += 1;
        copy_sources<<<source_blocks, source_threads>>>(
                d->fields[TM_EZFIELD],
                d->sources->x_source_position,
                d->sources->y_source_position,
                d->sources->source_type,
                d->sources->mean,
                d->sources->variance,
                d->sources->size,
                time_ticks);

        update_Hx<<<blocks, threads>>>(d->fields[TM_HXFIELD],
                                        d->fields[TM_EZFIELD],
                                        d->coefs[0],
                                        d->coefs[1]);

        update_Hy<<<blocks, threads>>>(d->fields[TM_HYFIELD],
                                        d->fields[TM_EZFIELD],
                                        d->coefs[0],
                                        d->coefs[1]);

        update_Ez<<<blocks, threads>>>(d->fields[TM_HXFIELD],
                                        d->fields[TM_HYFIELD],
                                        d->fields[TM_EZFIELD],
                                        d->coefs[2],
                                        d->coefs[3]);
    }
    float_to_color<<<blocks, threads>>> (d->output_bitmap,
                                        d->fields[TM_EZFIELD]);

    checkCudaErrors(hipMemcpy2D(bitmap->get_ptr(),
                                sizeof(float) * d->structure->x_index_dim,
                                d->output_bitmap,
                                d->structure->pitch,
                                sizeof(float) * d->structure->x_index_dim,
                                d->structure->y_index_dim,
                                hipMemcpyDeviceToHost));

    checkCudaErrors(hipEventRecord(d->stop, 0) );
    checkCudaErrors(hipEventSynchronize(d->stop));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, d->start, d->stop));
    d->totalTime +=elapsedTime;
    d->frames +=1;
    printf("Average time per frame: %3.1f ms\n", elapsedTime);
}


void tm_clear_memory_constants(Datablock *d){
    hipFree(d->constants[SIGMAINDEX]);
    hipFree(d->constants[SIGMA_STAR_INDEX]);
    hipFree(d->constants[EPSINDEX]);
    hipFree(d->constants[MUINDEX]);
}

void clear_memory_TM_simulation(Datablock *d){
    hipFree(d->fields[TM_EZFIELD]);
    hipFree(d->fields[TM_HYFIELD]);
    hipFree(d->fields[TM_HXFIELD]);
    hipFree(d->coefs[0]);
    hipFree(d->coefs[1]);
    hipFree(d->coefs[2]);
    hipFree(d->coefs[3]);
    hipFree(d->sources->x_source_position);
    hipFree(d->sources->y_source_position);
    hipFree(d->sources->source_type);
    hipFree(d->sources->mean);
    hipFree(d->sources->variance);
    checkCudaErrors(hipEventDestroy(d->start) );
    checkCudaErrors(hipEventDestroy(d->stop) );
}

size_t allocateTMMemory(Datablock *data, Structure structure){
    printf("The size of the structure is %ld", structure.size());
    size_t pitch;

    checkCudaErrors(hipMallocPitch( (void **) &data->output_bitmap,
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));

    checkCudaErrors(hipMallocPitch( (void **) &data->fields[TM_EZFIELD],
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));
    printf("%d\n", pitch);
    checkCudaErrors(hipMallocPitch( (void **) &data->fields[TM_HYFIELD],
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));
    checkCudaErrors(hipMallocPitch( (void **) &data->fields[TM_HXFIELD],
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));
    checkCudaErrors(hipMallocPitch( (void **) &data->constants[MUINDEX],
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));
    checkCudaErrors(hipMallocPitch( (void **) &data->constants[EPSINDEX],
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));
    checkCudaErrors(hipMallocPitch( (void **) &data->constants[SIGMAINDEX],
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));
    checkCudaErrors(hipMallocPitch( (void **) &data->constants[SIGMA_STAR_INDEX],
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));
    checkCudaErrors(hipMallocPitch( (void **) &data->coefs[0],
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));
    checkCudaErrors(hipMallocPitch( (void **) &data->coefs[1],
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));
    checkCudaErrors(hipMallocPitch( (void **) &data->coefs[2],
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));
    checkCudaErrors(hipMallocPitch( (void **) &data->coefs[3],
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));
    return pitch;
}

void initialize_TM_arrays(Datablock *data, Structure structure){
    int size = structure.grid_size();
    printf("%ld\n", size);
    printf("%ld\n", structure.x_index_dim);
    printf("%ld\n", structure.y_index_dim);

    // FIXME: Temporary fix for populating values.

    float * temp = (float *) malloc(structure.size());
    std::fill_n(temp, size, MU);
    checkCudaErrors(hipMemcpy2D(data->constants[MUINDEX], structure.pitch,
                temp, sizeof(float) * structure.x_index_dim,
                sizeof(float) * structure.x_index_dim,
                structure.y_index_dim,
                hipMemcpyHostToDevice));

    std::fill_n(temp, size, EPSILON * 20);
    checkCudaErrors(hipMemcpy2D(data->constants[EPSINDEX], structure.pitch,
                temp, sizeof(float) * structure.x_index_dim,
                sizeof(float) * structure.x_index_dim,
                structure.y_index_dim,
                hipMemcpyHostToDevice));

    std::fill_n(temp, size, 0.0);
    checkCudaErrors(hipMemcpy2D(data->constants[SIGMAINDEX], structure.pitch,
                temp, sizeof(float) * structure.x_index_dim,
                sizeof(float) * structure.x_index_dim,
                structure.y_index_dim,
                hipMemcpyHostToDevice));

    std::fill_n(temp, size, 0.0);
    checkCudaErrors(hipMemcpy2D(data->constants[SIGMA_STAR_INDEX], structure.pitch,
                temp, sizeof(float) * structure.x_index_dim,
                sizeof(float) *  structure.x_index_dim,
                structure.y_index_dim,
                hipMemcpyHostToDevice));

    // FIXME : For 2d pitch this has to be modified.
    dim3 blocks((data->structure->x_index_dim + BLOCKSIZE_X - 1) / BLOCKSIZE_X,
                (data->structure->y_index_dim + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y);
    dim3 threads(BLOCKSIZE_X, BLOCKSIZE_Y);


    initialize_array<<<blocks, threads>>>(data->fields[TM_HXFIELD], 0);
    initialize_array<<<blocks, threads>>>(data->fields[TM_HYFIELD], 0);
    initialize_array<<<blocks, threads>>>(data->fields[TM_EZFIELD], 0);
}

void copy_sources_device_to_host(HostSources * host_sources, DeviceSources *device_sources){
    int number_of_sources = host_sources->get_size();
    device_sources->size = number_of_sources;
    checkCudaErrors(hipMalloc((void**)&device_sources->x_source_position,
                number_of_sources * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&device_sources->y_source_position,
                number_of_sources * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&device_sources->source_type,
                number_of_sources * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&device_sources->mean,
                number_of_sources * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&device_sources->variance,
                number_of_sources * sizeof(float)));

    if(number_of_sources != 0){
    int *host_source_ptr = &(host_sources->x_source_position[0]);
    checkCudaErrors(hipMemcpy(device_sources->x_source_position, host_source_ptr,
                sizeof(int) * number_of_sources,hipMemcpyHostToDevice));


    host_source_ptr = &(host_sources->y_source_position[0]);
    checkCudaErrors(hipMemcpy(device_sources->y_source_position, host_source_ptr,
                sizeof(int) * number_of_sources,hipMemcpyHostToDevice));


    host_source_ptr = &(host_sources->source_type[0]);
    checkCudaErrors(hipMemcpy(device_sources->source_type, host_source_ptr,
                sizeof(int) * number_of_sources, hipMemcpyHostToDevice));

    float * mean_ptr = &(host_sources->mean[0]);
    checkCudaErrors(hipMemcpy(device_sources->mean, mean_ptr,
                sizeof(float) * number_of_sources, hipMemcpyHostToDevice));

    float * variance_ptr = &(host_sources->variance[0]);
    checkCudaErrors(hipMemcpy(device_sources->variance, variance_ptr,
                sizeof(float) * number_of_sources, hipMemcpyHostToDevice));
    }
}
