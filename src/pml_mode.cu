#include "pml_mode.h"

void anim_gpu_pml_tm(Datablock *d, int ticks){
    checkCudaErrors(hipEventRecord(d->start, 0) );
    dim3 blocks((d->structure->x_index_dim + BLOCKSIZE_X - 1) / BLOCKSIZE_X,
                (d->structure->y_index_dim + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y);
    dim3 threads(BLOCKSIZE_X, BLOCKSIZE_Y);

    dim3 source_threads(64, 1);
    dim3 source_blocks((d->sources->size + 63) / 64, 1);
    CPUAnimBitmap *bitmap = d->bitmap;
    static long time_ticks = 0;
    printf("time ticks = %ld", time_ticks);
    printf("time ticks = %ld", time_ticks);

    for(int i=0;i<100;i++){
        time_ticks += 1;
        copy_sources<<<source_blocks, source_threads>>>(
                d->fields[TM_PML_EZFIELD],
                d->sources->x_source_position,
                d->sources->y_source_position,
                d->sources->source_type,
                d->sources->mean,
                d->sources->variance,
                d->sources->size,
                time_ticks);

        update_Hx<<<blocks, threads>>>(d->fields[TM_PML_HXFIELD],
                                        d->fields[TM_PML_EZFIELD],
                                        d->coefs[0],
                                        d->coefs[1]);

        update_Hy<<<blocks, threads>>>(d->fields[TM_PML_HYFIELD],
                                        d->fields[TM_PML_EZFIELD],
                                        d->coefs[2],
                                        d->coefs[3]);

        update_pml_ezx<<<blocks, threads>>>(d->fields[TM_PML_EZXFIELD],
                                            d->fields[TM_PML_HYFIELD],
                                            d->coefs[4],
                                            d->coefs[5]);

        update_pml_ezy<<<blocks, threads>>>(d->fields[TM_PML_EZYFIELD],
                                            d->fields[TM_PML_HXFIELD],
                                            d->coefs[6],
                                            d->coefs[7]);

        update_pml_ez<<<blocks, threads>>>(d->fields[TM_PML_EZXFIELD],
                                            d->fields[TM_PML_EZYFIELD],
                                            d->fields[TM_PML_EZFIELD]);
    }

    float_to_color<<<blocks, threads>>> (d->output_bitmap,
                                        d->fields[TM_PML_EZFIELD]);

    checkCudaErrors(hipMemcpy(bitmap->get_ptr(), d->output_bitmap,
                        bitmap->image_size(), hipMemcpyDeviceToHost));

    checkCudaErrors(hipEventRecord(d->stop, 0) );
    checkCudaErrors(hipEventSynchronize(d->stop));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, d->start, d->stop));
    d->totalTime +=elapsedTime;
    d->frames +=1;
    printf("Average time per frame: %3.1f ms\n", elapsedTime);
}

void tm_pml_clear_memory_constants(Datablock *d){
    hipFree(d->constants[SIGMAINDEX_X]);
    hipFree(d->constants[SIGMAINDEX_Y]);
    hipFree(d->constants[SIGMA_STAR_INDEX_X]);
    hipFree(d->constants[SIGMA_STAR_INDEX_Y]);
    hipFree(d->constants[EPSINDEX]);
    hipFree(d->constants[MUINDEX]);
}

void clear_memory_TM_PML_simulation(Datablock *d){
    hipFree(d->fields[TM_PML_EZFIELD]);
    hipFree(d->fields[TM_PML_EZXFIELD]);
    hipFree(d->fields[TM_PML_EZYFIELD]);
    hipFree(d->fields[TM_HYFIELD]);
    hipFree(d->fields[TM_HXFIELD]);
    hipFree(d->constants[SIGMAINDEX]);
    hipFree(d->constants[SIGMA_STAR_INDEX]);
    hipFree(d->constants[EPSINDEX]);
    hipFree(d->constants[MUINDEX]);
    hipFree(d->coefs[0]);
    hipFree(d->coefs[1]);
    hipFree(d->coefs[2]);
    hipFree(d->coefs[3]);
    hipFree(d->coefs[4]);
    hipFree(d->coefs[5]);
    hipFree(d->coefs[6]);
    hipFree(d->coefs[7]);
    hipFree(d->sources->x_source_position);
    hipFree(d->sources->y_source_position);
    hipFree(d->sources->source_type);
    hipFree(d->sources->mean);
    hipFree(d->sources->variance);
    checkCudaErrors(hipEventDestroy(d->start) );
    checkCudaErrors(hipEventDestroy(d->stop) );
}

void tm_pml_allocate_memory(Datablock *data, Structure structure){

    checkCudaErrors(hipMalloc( (void **) &data->output_bitmap,
                    structure.size()));
    for(int i = 0;i < 5;i++){
    checkCudaErrors(hipMalloc( (void **) &data->fields[i],
                    structure.size() ));
    }

    for(int i = 0; i < 6; i++){
    checkCudaErrors(hipMalloc( (void **) &data->constants[i],
                    structure.size() ));
    }

    for(int i = 0;i < 8; i++){
    checkCudaErrors(hipMalloc( (void **) &data->coefs[i],
                    structure.size() ));
    }

}

void tm_pml_initialize_arrays(Datablock *data, Structure structure){
    int size = structure.grid_size();
    printf("%ld\n", size);
    printf("%ld\n", structure.x_index_dim);
    printf("%ld\n", structure.y_index_dim);

    // FIXME: Temporary fix for populating values.

    float * temp = (float *) malloc(structure.size());
    std::fill_n(temp, size, MU);
    hipMemcpy(data->constants[MUINDEX],temp,structure.size(),
                hipMemcpyHostToDevice);

    std::fill_n(temp, size, EPSILON * 20);
    hipMemcpy(data->constants[EPSINDEX],temp,structure.size(),
                hipMemcpyHostToDevice);

    std::fill_n(temp, size, 0.0);
    hipMemcpy(data->constants[SIGMAINDEX_X],temp,structure.size(),
                hipMemcpyHostToDevice);

    std::fill_n(temp, size, 0.0);
    hipMemcpy(data->constants[SIGMAINDEX_Y],temp,structure.size(),
                hipMemcpyHostToDevice);

    std::fill_n(temp, size, 0.0);
    hipMemcpy(data->constants[SIGMA_STAR_INDEX_X],temp,structure.size(),
                hipMemcpyHostToDevice);

    std::fill_n(temp, size, 0.0);
    hipMemcpy(data->constants[SIGMA_STAR_INDEX_Y],temp,structure.size(),
                hipMemcpyHostToDevice);

    thrust::device_ptr<float> hx_field_ptr(data->fields[TM_PML_HXFIELD]);
    thrust::fill(hx_field_ptr, hx_field_ptr + size, 0);

    thrust::device_ptr<float> hy_field_ptr(data->fields[TM_PML_HYFIELD]);
    thrust::fill(hy_field_ptr, hy_field_ptr + size, 0);

    thrust::device_ptr<float> ez_field_ptr(data->fields[TM_PML_EZFIELD]);
    thrust::fill(ez_field_ptr, ez_field_ptr + size, 0);

    thrust::device_ptr<float> ezx_field_ptr(data->fields[TM_PML_EZXFIELD]);
    thrust::fill(ezx_field_ptr, ezx_field_ptr + size, 0);

    thrust::device_ptr<float> ezy_field_ptr(data->fields[TM_PML_EZYFIELD]);
    thrust::fill(ezy_field_ptr, ezy_field_ptr + size, 0);
}
